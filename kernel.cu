#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <limits.h>
#include <stdlib.h>
#include <inttypes.h>
#include <time.h>
#include "Simon.h"

// Cipher Operation Macros
#define shift_one ((x_word << 1) | (x_word >> (word_size - 1)))
#define shift_eight ((x_word << 8) | (x_word >> (word_size - 8)))
#define shift_two ((x_word << 2) | (x_word >> (word_size - 2)))
#define rshift_three(x) (x >> 3) |((x & 0x7) << (word_size - 3))
#define rshift_one(x)   (x >> 1) |((x & 0x1) << (word_size - 1))

uint64_t z_arrays[5] = { 0x19C3522FB386A45F, 0x16864FB8AD0C9F71, 0x3369F885192C0EF5, 0x3C2CE51207A635DB, 0x3DC94C3A046D678B };

// Valid Cipher Parameters
const uint8_t simon_rounds[] = { 32, 36, 36, 42, 44, 52, 54, 68, 69, 72 };
const uint8_t simon_block_sizes[] = { 32, 48, 48, 64, 64, 96, 96, 128, 128, 128 };
const uint16_t simon_key_sizes[] = { 64, 72, 96, 96, 128, 96, 144, 128, 192, 256 };
const uint8_t  z_assign[] = { 0, 0, 1, 2, 3, 2, 3, 2, 3, 4 };

uint8_t Simon_Init(Simon_Cipher *cipher_object, enum simon_cipher_config_t cipher_cfg, enum mode_t c_mode, uint8_t *key, uint8_t *iv, uint8_t *counter) {

	if (cipher_cfg != Simon_128_128) {
		return -1;
	}

	cipher_object->block_size = simon_block_sizes[cipher_cfg];
	cipher_object->key_size = simon_key_sizes[cipher_cfg];
	cipher_object->round_limit = simon_rounds[cipher_cfg];
	cipher_object->cipher_cfg = cipher_cfg;
	cipher_object->z_seq = z_assign[cipher_cfg];
	uint8_t word_size = simon_block_sizes[cipher_cfg] >> 1;
	uint8_t word_bytes = word_size >> 3;
	uint8_t key_words = simon_key_sizes[cipher_cfg] / word_size;
	uint64_t sub_keys[4] = {};
	uint64_t mod_mask = ULLONG_MAX >> (64 - word_size);

	// Setup
	for (int i = 0; i < key_words; i++) {
		memcpy(&sub_keys[i], key + (word_bytes * i), word_bytes);
	}

	uint64_t tmp1, tmp2;
	uint64_t c = 0xFFFFFFFFFFFFFFFC;

	// Store First Key Schedule Entry
	memcpy(cipher_object->key_schedule, &sub_keys[0], word_bytes);

	for (int i = 0; i < simon_rounds[cipher_cfg] - 1; i++) {
		tmp1 = rshift_three(sub_keys[key_words - 1]);

		if (key_words == 4) {
			tmp1 ^= sub_keys[1];
		}

		tmp2 = rshift_one(tmp1);
		tmp1 ^= sub_keys[0];
		tmp1 ^= tmp2;

		tmp2 = c ^ ((z_arrays[cipher_object->z_seq] >> (i % 62)) & 1);

		tmp1 ^= tmp2;

		// Shift Sub Words
		for (int j = 0; j < (key_words - 1); j++){
			sub_keys[j] = sub_keys[j + 1];
		}
		sub_keys[key_words - 1] = tmp1 & mod_mask;

		// Append sub key to key schedule
		memcpy(cipher_object->key_schedule + (word_bytes * (i + 1)), &sub_keys[0], word_bytes);
	}

	return 0;
}

void Simon_Encrypt_128(Simon_Cipher *cipher_object, uint8_t *plaintext, uint8_t *ciphertext) {

	const uint8_t word_size = 64;
	uint64_t y_word = *(uint64_t *)plaintext;
	uint64_t x_word = *(((uint64_t *)plaintext) + 1);
	uint64_t *round_key_ptr = (uint64_t *)cipher_object->key_schedule;
	uint64_t *word_ptr = (uint64_t *)ciphertext;

	for (uint8_t i = 0; i < cipher_object->round_limit; i++) {  // Block size 32 has only one round number option

		// Shift, AND , XOR ops
		uint64_t temp = (shift_one & shift_eight) ^ y_word ^ shift_two;

		// Feistel Cross
		y_word = x_word;

		// XOR with Round Key
		x_word = temp ^ *(round_key_ptr + i);
		//printf("x: %d, %"PRIx64"\n", i, x_word);
	}
	// Assemble Ciphertext Output Array   
	*word_ptr = y_word;
	*(word_ptr + 1) = x_word;
}

__global__ void Simon_Decrypt_128(Simon_Cipher *cipher_object, uint8_t* keySchedule, uint8_t *ciphertext, uint8_t *plaintext)
{
	const uint8_t word_size = 64;
	uint64_t x_word = *(uint64_t *)ciphertext;
	uint64_t y_word = *(((uint64_t *)ciphertext) + 1);
	uint64_t *round_key_ptr = (uint64_t *)(keySchedule);
	uint64_t *word_ptr = (uint64_t *)plaintext;

	//printf("x_word %02x\n", x_word);
	//printf("y_word %02x\n", y_word);

	//printf("\n\n");

	//printf("round_limit %d\n", cipher_object->round_limit);
	int8_t round_limit_minus_one = cipher_object->round_limit - 1;

	for (int8_t i = round_limit_minus_one; i >= 0; i--) {

		//printf(" x: %d, %"PRIx64"\n", i, x_word);
		//printf("y: %d, %"PRIx64"\n", i, y_word);

		// Shift, AND , XOR ops
		//uint64_t temp = (shift_one & shift_eight) ^ y_word ^ shift_two;
		uint64_t temp = (((x_word << 1) | (x_word >> (word_size - 1))) &
			((x_word << 8) | (x_word >> (word_size - 8)))) ^ y_word ^
			((x_word << 2) | (x_word >> (word_size - 2)));
		//printf("temp %02d: %"PRIx64"\n", i, temp);

		// Feistel Cross
		y_word = x_word;
		//printf("x_word, y_word after (y_word = x_word) %02d: %02x, %02x\n", i, x_word, y_word);

		// XOR with Round Key
		x_word = temp ^ *(round_key_ptr + i);
	}

	// Assemble Plaintext Output Array   
	*word_ptr = x_word;
	*(word_ptr + 1) = y_word;

	return;
}

int main(void) {
	clock_t start = clock();

	// Create reuseable cipher objects for each alogirthm type
	Simon_Cipher my_simon_cipher = *(Simon_Cipher *)malloc(sizeof(Simon_Cipher));

	// Create generic tmp variables
	uint8_t ciphertext_buffer[16];

	uint8_t deciphertext_buffer[16];
	uint32_t result;

	// Initialize IV and Counter Values for Use with Block Modes
	uint8_t my_IV[] = { 0x32, 0x14, 0x76, 0x58 };
	uint8_t my_counter[] = { 0x2F, 0x3D, 0x5C, 0x7B };
	Simon_Cipher *d_my_simon_cipher;
	uint8_t *d_simon128_128_plain, *d_ciphertext_buffer, *d_simon128_128_cipher;
	uint8_t *d_key_schedule;

	printf("***********************************\n");
	printf("******* Simon Cipher Tests ********\n");
	printf("***********************************\n");

	// Simon 128/128 Test
	// Key: 0f0e0d0c0b0a0908 0706050403020100 Plaintext: 6373656420737265 6c6c657661727420 Ciphertext: 49681b1e1e54fe3f 65aa832af84e0bbc
	printf("Test Simon 128/128\n");
	uint8_t simon128_128_key[] = { 0x00, 0x01, 0x02, 0x03, 0x04, 0x05, 0x06, 0x07, 0x08, 0x09, 0x0A, 0x0B, 0x0C, 0x0D, 0x0E, 0x0F };
	uint8_t simon128_128_plain[] = { 0x21, 0x75, 0x6a, 0x84, 0x76, 0x65, 0x6c, 0x6c, 0x65, 0x72, 0x73, 0x20, 0x90, 0x88, 0x73, 0x63 };
	uint8_t simon128_128_cipher[16];
	result = Simon_Init(&my_simon_cipher, Simon_128_128, ECB, simon128_128_key, my_IV, my_counter);

	printf("\nEncryption Test:\n");

	Simon_Encrypt_128(&my_simon_cipher, simon128_128_plain, ciphertext_buffer);

	for (int i = 0; i < 16; i++) {
		printf("Ciphertext Byte %02d: %02x - %02x", i, simon128_128_plain[i], ciphertext_buffer[i]);
		simon128_128_cipher[i] = ciphertext_buffer[i];
		printf("\n");
	}

	hipMalloc((Simon_Cipher**)&d_my_simon_cipher, sizeof(Simon_Cipher));
	hipMalloc((void **)&d_simon128_128_cipher, sizeof(uint8_t) * sizeof(simon128_128_cipher));
	hipMalloc((void **)&d_ciphertext_buffer, sizeof(uint8_t) * sizeof(ciphertext_buffer));
	hipMalloc((void **)&d_key_schedule, sizeof(uint8_t) * sizeof(my_simon_cipher.key_schedule)); // 576 is size of key_schedule array

	if (0 != hipMemcpy(d_my_simon_cipher, &my_simon_cipher, sizeof(Simon_Cipher), hipMemcpyHostToDevice)) {
		printf("Mem allocation error n");
		getchar();
	}

	if (0 != hipMemcpy(d_simon128_128_cipher, simon128_128_cipher, sizeof(uint8_t) * sizeof(simon128_128_cipher), hipMemcpyHostToDevice)) {
		printf("Mem allocation error n");
		getchar();
	}

	if (0 != hipMemcpy(d_ciphertext_buffer, ciphertext_buffer, sizeof(uint8_t) * sizeof(ciphertext_buffer), hipMemcpyHostToDevice)) {
		printf("Mem allocation error n");
		getchar();
	}

	hipMemcpy(d_key_schedule, my_simon_cipher.key_schedule, sizeof(uint8_t) *  sizeof(my_simon_cipher.key_schedule), hipMemcpyHostToDevice);

	Simon_Decrypt_128 <<<1, 1>>>(d_my_simon_cipher, d_key_schedule, d_simon128_128_cipher, d_ciphertext_buffer);
	hipDeviceSynchronize();

	hipMemcpy(&my_simon_cipher, d_my_simon_cipher, sizeof(Simon_Cipher), hipMemcpyDeviceToHost);
	hipMemcpy(simon128_128_cipher, d_simon128_128_cipher, sizeof(uint8_t) * sizeof(simon128_128_cipher), hipMemcpyDeviceToHost);
	hipMemcpy(ciphertext_buffer, d_ciphertext_buffer, sizeof(uint8_t) * sizeof(ciphertext_buffer), hipMemcpyDeviceToHost);

	printf("\nDecryption Test:\n");
	for (int i = 0; i < 16; i++) {
		printf("Plaintext Byte %02d: %02x", i, ciphertext_buffer[i]);
		if (ciphertext_buffer[i] != simon128_128_plain[i]) printf("  FAIL\n");
		else printf("\n");
	}
	printf("\n");

	hipFree(d_my_simon_cipher);
	hipFree(d_ciphertext_buffer);
	hipFree(d_simon128_128_cipher);

	clock_t stop = clock();

	printf("Time Taken: %f seconds\n", ((double)stop - start) / CLOCKS_PER_SEC);

	system("pause");

	return 0;
}